
#include <hip/hip_runtime.h>
#include <stdint.h>


// Kernel de minera��o para GPUs
__device__ void sha256_double(const uint8_t* header, uint32_t nonce, uint8_t* hash) {
    for (int i = 0; i < 32; ++i) {
        hash[i] = header[i % 80] ^ (nonce >> (i % 8));
    }
}

__global__ void StartMiner(uint8_t* header, uint32_t start_nonce, uint32_t* result, uint32_t target) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x + start_nonce;
    uint8_t hash[32];

    sha256_double(header, idx, hash);

    if (hash[0] == 0 && hash[1] == 0 && hash[2] < target) {
        *result = idx;
    }
}
